#include <stdio.h>
#include <stdlib.h>

#include <sys/types.h>
#include <sys/ipc.h>
#include <sys/shm.h>

#include <string.h>

#include "hipcub/hipcub.hpp"

int main(int argc, char *argv[]) {
	float lower_level = atof(argv[1]);
	float upper_level = atof(argv[2]);

	if (lower_level > upper_level) {
		printf("FAIL: lower level is bigger than upper level\n");
		exit(1);
	}

	int num_bins = atoi(argv[3]);
	int num_levels = num_bins + 1;
	int num_samples = atoi(argv[4]);

	int shmID_matBuf1 = atoi(argv[5]);
	int shmID_histogram = atoi(argv[6]);
	
	float* pBase_matBuf1;
	int* pBase_histogram;

	if ((pBase_matBuf1 = (float*) shmat(shmID_matBuf1, NULL, 0)) == (float*) -1) {
		printf("FAIL: cannot get SHM\n");
		exit(1);
	}

	if ((pBase_histogram = (int*) shmat(shmID_histogram, NULL, 0)) == (int*) -1) {
		printf("FAIL: cannot get SHM\n");
		exit(1);
	}

	float* d_matBuf1;
	int* d_histogram;

	hipMalloc(&d_matBuf1, num_samples * sizeof(float));
	hipMemcpy(d_matBuf1, pBase_matBuf1, num_samples * sizeof(float), hipMemcpyHostToDevice);

	hipMalloc(&d_histogram, num_bins * sizeof(int));

	void* d_temp_storage = NULL;
	size_t temp_storage_bytes = 0;

	hipcub::DeviceHistogram::HistogramEven(d_temp_storage, temp_storage_bytes, d_matBuf1, d_histogram, num_levels, lower_level, upper_level, num_samples);

	hipMalloc(&d_temp_storage, temp_storage_bytes);

	hipcub::DeviceHistogram::HistogramEven(d_temp_storage, temp_storage_bytes, d_matBuf1, d_histogram, num_levels, lower_level, upper_level, num_samples);

	hipMemcpy(pBase_histogram, d_histogram, (num_bins) * sizeof(int), hipMemcpyDeviceToHost);

	return 0;
}
